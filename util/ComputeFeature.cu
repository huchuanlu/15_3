#include "hip/hip_runtime.h"
#include "ComputeFeature.hpp"
#include <stdio.h>
#include <limits.h>
//dim3 GridDim(int num_blocks) {
//  dim3 grid_dim;
//  if (num_blocks <= GRID_DIM_X) {
//    grid_dim.x = num_blocks; 
//  } else if(num_blocks <= GRID_DIM_X * GRID_DIM_Y) {
//    grid_dim.x = GRID_DIM_X;
//    grid_dim.y = num_blocks / GRID_DIM_X + 1;
//  } else if(num_blocks <= GRID_DIM_X * GRID_DIM_Y * GRID_DIM_Z) {
//    grid_dim.x = GRID_DIM_X;
//    grid_dim.y = GRID_DIM_Y;
//    grid_dim.z = num_blocks / (GRID_DIM_X * GRID_DIM_Y) + 1;
//  }
//  return grid_dim;
//}
__device__ unsigned int float_to_uint(float f)
{
  unsigned int *p = reinterpret_cast<unsigned int*>(&f);
  unsigned int input = *p;
  unsigned int mask = -int(input  >> 31) | 0x80000000;
  return input ^ mask;
}
__device__ float uint_to_float(const unsigned int f)
{
  unsigned int mask = ((f >> 31) - 1) | 0x80000000;
  unsigned int output = f ^ mask;
  float *p = reinterpret_cast<float*>(&output);
  return *p;
}
__global__ void compute_pr(float* local_fea, float* reg_area,  float local_map_sum, const int num_reg, const int num_local_fea) {
  int reg_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (reg_id < num_reg) {
    float *cur_local_fea = local_fea + reg_id * num_local_fea;
    float intersection_local = cur_local_fea[0];
    float union_area_local = cur_local_fea[1];
    float reg_area_local = reg_area[reg_id]; 

    cur_local_fea[0] = intersection_local / reg_area_local;
    cur_local_fea[1] = intersection_local / local_map_sum; 
    cur_local_fea[2] = cur_local_fea[0] * cur_local_fea[1]; 
    cur_local_fea[3] = intersection_local / union_area_local;
  }
} 


__global__ void compute_local_saliency(const bool *local_map, const bool *reg, const bool compute_reg_area, const int width, const int height, const int num_reg, const int num_local_fea, const float local_map_sum, const int blocks_per_reg, float *local_fea, float *reg_area) {
  // Step 1: compute the index of current threads and determine whether it is on a region proposal
  int reg_id = blockIdx.x / blocks_per_reg;
  int px_id = (blockIdx.x % blocks_per_reg) * blockDim.x + threadIdx.x;
  int map_offset = width * height;
  const bool *cur_reg = reg + reg_id * map_offset;
  float *cur_local_fea = local_fea + num_local_fea * reg_id;

  // Step 2: init the shared memories: intersection, union_area, reg_area 
  //__shared__ float intersection_tmp;
  //__shared__ float union_area_tmp;
  //__shared__ float reg_area_tmp;
  extern __shared__ float shared_local_saliency[];
  float reg_area_tmp;
  // Compute reg_are_tmp
  if (compute_reg_area) {
    if (px_id < map_offset && cur_reg[px_id]) {
      shared_local_saliency[threadIdx.x] = 1; 
    } else {
      shared_local_saliency[threadIdx.x] = 0;
    }
    __syncthreads();
      for (int s = (blockDim.x + 1) / 2, e = blockDim.x / 2; s > 0; s = (s + 1) / 2, e /= 2) {
	if (threadIdx.x < e || threadIdx.x == 0) {
	  atomicAdd(&shared_local_saliency[threadIdx.x], shared_local_saliency[threadIdx.x + s]); 
	}
	__syncthreads();
	if (s == 1) break;
      }
    reg_area_tmp = shared_local_saliency[0];
  }
  // Compute intersection_tmp 
  if (px_id < map_offset && cur_reg[px_id] && local_map[px_id]) {
    shared_local_saliency[threadIdx.x] = 1; 
  } else {
    shared_local_saliency[threadIdx.x] = 0;
  }
  __syncthreads();
    for (int s = (blockDim.x + 1) / 2, e = blockDim.x / 2; s > 0; s = (s + 1) / 2, e /= 2) {
      if (threadIdx.x < e || threadIdx.x == 0) {
	atomicAdd(&shared_local_saliency[threadIdx.x], shared_local_saliency[threadIdx.x + s]); 
      }
      if (s == 1) break;
      __syncthreads();
    }
  float intersection_tmp = shared_local_saliency[0];
  // Compute union_area_tmp 
  if (px_id < map_offset && (cur_reg[px_id] || local_map[px_id])) {
    shared_local_saliency[threadIdx.x] = 1; 
  } else {
    shared_local_saliency[threadIdx.x] = 0;
  }
  __syncthreads();

  for (int s = (blockDim.x + 1) / 2, e = blockDim.x / 2; s > 0; s = (s + 1) / 2, e /= 2) {
    if (threadIdx.x < e || threadIdx.x == 0) {
      atomicAdd(&shared_local_saliency[threadIdx.x], shared_local_saliency[threadIdx.x + s]); 
    }
    __syncthreads();
    if (s ==1) break;
  }
  float union_area_tmp = shared_local_saliency[0];



  // Step 4: sum the block shared memories to global momories
  if (threadIdx.x == 0) {
    atomicAdd(cur_local_fea, intersection_tmp); // 0: intersetcion
    atomicAdd(&cur_local_fea[1], union_area_tmp); // 1: union_area_tmp
    if(compute_reg_area) { atomicAdd(&reg_area[reg_id], reg_area_tmp); } // 2: reg_area
  }
  __syncthreads();

  //if (px_id < map_offset) {
  //  if(px_id == 0) {
  //    //float intersection_local = cur_local_fea[0];
  //    //float union_area_local = cur_local_fea_space[1];
  //    //float reg_area_local = compute_reg_area ? cur_local_fea[2] : reg_area[reg_id]; 
  //    //float intersection_local, union_area_local, reg_area_local;
  //    //intersection_local = atomicExch(cur_local_fea, (float)0);
  //    //union_area_local = atomicExch(&cur_local_fea[1],(float)0);
  //    //reg_area_local = atomicExch(&cur_local_fea[2], (float)0);
  //    //__syncthreads();

  //    //cur_local_fea[0] = intersection_local / reg_area_local;
  //    //cur_local_fea[1] = intersection_local / local_map_sum; 
  //    //cur_local_fea[2] = cur_local_fea[0] * cur_local_fea[1]; 
  //    //cur_local_fea[3] = intersection_local / union_area_local;
  //    //cur_local_fea[0] = intersection_local;//intersection_local;
  //    //cur_local_fea[1] = union_area_local;
  //    //cur_local_fea[2] = intersection_local;
  //    //cur_local_fea[3] = intersection_local / union_area_local;
  //    //if(compute_reg_area) reg_area[reg_id] = reg_area_local;
  //  }
  //}
  //__syncthreads();
}

__global__ void  compute_color_hist(const int* qualified_fea, const bool* reg, const float* reg_area, const int width, const int height, const int num_reg, const int num_fea, const int num_bin,  const int blocks_per_reg, float* hist) {
  // Step 1: parameter initialziation and compute the index of current threads and determine whether it is on a region proposal
  int shared_hist_size = num_bin;
  extern __shared__ float shared_hist[];
  //  __shared__ unsigned int shared_hist[12288];
  for (int i = threadIdx.x; i < shared_hist_size; i += blockDim.x) 
    shared_hist[i] = 0;
  __syncthreads();

  int reg_id = blockIdx.x / blocks_per_reg;
  int px_id = (blockIdx.x % blocks_per_reg) * blockDim.x + threadIdx.x;
  int fea_id = blockIdx.y;
  int im_offset = width * height;
  int hist_offset = num_bin;
  const bool * cur_reg = reg + reg_id * im_offset;
  if (px_id < im_offset && cur_reg[px_id]) {
    // Step 2:  compute hist in each block 
    //for (int fea_id = 0; fea_id < num_fea; fea_id++) {
    const int* cur_fea = qualified_fea + im_offset * fea_id;
    // unsigned int * cur_shared_hist = shared_hist + hist_offset * fea_id;
    //   if (cur_reg[px_id]) { 
    atomicAdd(&shared_hist[cur_fea[px_id]], 1);
    //   }
    // }
  }
  __syncthreads();
  // Step 3: collect the final hist from each block
  for (int i = threadIdx.x; i < shared_hist_size; i += blockDim.x) {
    float *cur_hist = hist + (reg_id * num_fea + fea_id) * hist_offset + i;
    float cur_shared_hist_value = shared_hist[i] / reg_area[reg_id];
    atomicAdd(cur_hist, cur_shared_hist_value);
  }
  __syncthreads();
}

__global__ void compute_color_mean(const float* fea, const bool* reg, const float* reg_area, const int width, const int height, const int num_reg, const int num_fea, const int blocks_per_reg, float* fea_mean) {
  // Step 1: parameter initialziation and compute the index of current threads and determine whether it is on a region proposal
  int shared_mean_size = COLOR_CHANNEL * num_fea;
  extern __shared__ float shared_mean[];
  for (int i = threadIdx.x; i < shared_mean_size; i += blockDim.x) {
    shared_mean[i] = 0;
  }
  __syncthreads();

  int reg_id = blockIdx.x / blocks_per_reg;
  int px_id = (blockIdx.x % blocks_per_reg) * blockDim.x + threadIdx.x;
  int im_offset = width * height;
  const bool * cur_reg = reg + reg_id * im_offset;
  // Step 2: compute mean values within blocks
  if (px_id < im_offset && cur_reg[px_id]) {
    for (int fea_id = 0; fea_id < shared_mean_size; fea_id++) {
      const float *cur_fea = fea + fea_id * im_offset;
      float *cur_shared_mean = shared_mean + fea_id;
      atomicAdd(cur_shared_mean, cur_fea[px_id]);
    }
  }
  __syncthreads();
  // Step 3: Average within block
  for (int i = threadIdx.x; i < shared_mean_size; i += blockDim.x) {
    shared_mean[i] /= reg_area[reg_id];
  }
  __syncthreads();
  // Step 4: collect mean values from shared memories
  float *cur_fea_mean = fea_mean + reg_id * shared_mean_size;
  for (int fea_id = threadIdx.x; fea_id < shared_mean_size; fea_id += blockDim.x) {
    atomicAdd(&cur_fea_mean[fea_id], shared_mean[fea_id]);
  }
  __syncthreads();
}

__global__ void compute_color_var(const float* fea, const float* fea_mean, const bool* reg, const float* reg_area,  const int width, const int height, const int num_reg, const int num_fea, const int blocks_per_reg, float* fea_var) {
  // Step 1: parameter initialziation and compute the index of current threads and determine whether it is on a region proposal
  int shared_var_size = COLOR_CHANNEL * num_fea;
  extern __shared__ float shared_var[];
  for (int i = threadIdx.x; i < shared_var_size; i += blockDim.x) {
    shared_var[i] = 0;
  }
  __syncthreads();

  int reg_id = blockIdx.x / blocks_per_reg;
  int px_id = (blockIdx.x % blocks_per_reg) * blockDim.x + threadIdx.x;
  int im_offset = width * height;
  const bool * cur_reg = reg + reg_id * im_offset;
  // Step 2: compute mean values within blocks
  if (px_id < im_offset && cur_reg[px_id]) {
    for (int fea_id = 0; fea_id < shared_var_size; fea_id++) {
      const float *cur_fea = fea + fea_id * im_offset;
      float *cur_shared_var = shared_var + fea_id;
      const float *cur_fea_mean = fea_mean + reg_id * shared_var_size;
      float deviation = cur_fea[px_id] - cur_fea_mean[fea_id];
      atomicAdd(cur_shared_var, deviation * deviation);
    }
  }
  __syncthreads();
  // Step 3: average within block
  for (int i = threadIdx.x; i < shared_var_size; i += blockDim.x) {
    shared_var[i] /= (reg_area[reg_id] - 1);
  }
  // Step 4: collect var from shared memories to device memories
  float *cur_fea_var = fea_var + reg_id * shared_var_size;
  for (int fea_id = threadIdx.x ; fea_id < shared_var_size; fea_id += blockDim.x) {
    atomicAdd(&cur_fea_var[fea_id], shared_var[fea_id]);
  }
  __syncthreads();
}

__global__ void compute_geo_fea(const bool* reg, const float* reg_area, const int width, const int height, const int num_reg, const int blocks_per_reg, float* geo_fea) {
  // Step 1: compute pixel location and region id, and initialize shared memory 
  int reg_id = blockIdx.x / blocks_per_reg;
  int px_id = (blockIdx.x % blocks_per_reg) * blockDim.x + threadIdx.x;
  int im_offset = width * height;
  const bool * cur_reg = reg + reg_id * im_offset;
  __shared__ float center_x, center_y;
  __shared__ int max_x, max_y, min_x, min_y;
  center_x = 0; center_y = 0; max_x = 0; min_x = width; max_y = 0; min_y = height;
  // Step 2: accumulate coordinates and record extrema within each block
  if (px_id < im_offset && cur_reg[px_id]) {
    int x = px_id / height + 1;
    int y = px_id % height + 1;
    float normalized_x = x / (1.0 * reg_area[reg_id]);
    float normalized_y = y / (1.0 * reg_area[reg_id]);
    atomicAdd(&center_x, normalized_x);
    atomicAdd(&center_y, normalized_y);
    atomicMax(&max_x, x);
    atomicMax(&max_y, y);
    atomicMin(&min_x, x);
    atomicMin(&min_y, y);
  }
  __syncthreads();
  // Step 3: collect results from blocks to device memories
  int *cur_geo_fea_int = reinterpret_cast<int*>(geo_fea + reg_id * NUM_GEO_FEA);
  float *cur_geo_fea_float = geo_fea + reg_id * NUM_GEO_FEA;
  // init global values:
  if (px_id == 0)
  {
    cur_geo_fea_int[2] = INT_MAX;
    cur_geo_fea_int[3] = INT_MAX;
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    atomicAdd(&cur_geo_fea_float[0], center_x);
    // cur_geo_fea_float[0] = 100;
  }
  if (threadIdx.x == 1) {
    atomicAdd(&cur_geo_fea_float[1], center_y);
  }
  if (threadIdx.x == 2) {
    atomicMin(&cur_geo_fea_int[2], min_x);
  }
  if (threadIdx.x == 3) {
    atomicMin(&cur_geo_fea_int[3],min_y);
  }
  if (threadIdx.x == 4) {
    atomicMax(&cur_geo_fea_int[4], max_x);
  }
  if (threadIdx.x == 5) {
    atomicMax(&cur_geo_fea_int[5], max_y);
  }
  if(threadIdx.x == 7) {
    cur_geo_fea_float[7] = reg_area[reg_id];
  }
  __syncthreads();
  // Convert unsinged int to float
}
// center_x, center_y, min_x, min_y, max_x, max_y, reg_area

__global__ void process_geo_fea(float* geo_fea, const bool* reg, const int num_reg, const int width, const int height) {
  int shared_size = max(width, height);
  //int half_shared_size = shared_size / 2;
  int thread_id = threadIdx.x;
  int reg_id = blockIdx.x;
  extern __shared__ int shared_space[]; 

  float width_float = static_cast<float>(width);
  float height_float = static_cast<float>(height);
  int *cur_geo_fea_int = reinterpret_cast<int*>(geo_fea + reg_id * NUM_GEO_FEA);
  float *cur_geo_fea_float = geo_fea + reg_id * NUM_GEO_FEA;
  const bool *cur_reg = reg + width * height * reg_id;
  float center_x = cur_geo_fea_float[0];
  float center_y = cur_geo_fea_float[1];
  int center_x_int = static_cast<int>(center_x)-1;
  int center_y_int = static_cast<int>(center_y)-1;
  float normalized_center_x = center_x / width_float;
  float normalized_center_y = center_y / height_float;
  float normalized_min_x = cur_geo_fea_int[2] / width_float;
  float normalized_min_y = cur_geo_fea_int[3] / height_float;
  float normalized_max_x = cur_geo_fea_int[4] / width_float;
  float normalized_max_y = cur_geo_fea_int[5] / height_float;
  int id_left = center_y_int; // the index of the left most pixel on the same raw of center
  int id_top = center_x_int * height; // the index of the top most pixel on the same col of center
  __syncthreads();

  //shared_space to compute min_center_x
  for (int i = thread_id; i < width; i += blockDim.x) { // shared_space == blockDim.x
    if (cur_reg[id_left + i * height]) {
      shared_space[i] = i + 1; // store x cooridnates
    } else {
      shared_space[i] = width; // if the pixel not on the region, set the maximum coordinate
    }
  }
  __syncthreads();
  // Continue after syncthreads
  // peform reduction to compute minimum x coordinates
  for (int s = (width + 1) / 2, e = width / 2; s > 0; s = (s + 1) / 2, e /= 2) {
    if (thread_id < e || thread_id == 0) {
      atomicMin(&shared_space[thread_id], shared_space[thread_id + s]);
    }
    __syncthreads();
    if (s == 1) break;
  }

  float min_center_x;
  if (thread_id == 0) {
    min_center_x = shared_space[0] / width_float;
  }

  //////////////////////////////////////////////////////////////////
  // reload shared_space to compute max_center_x
  for (int i = thread_id; i < width; i += blockDim.x) {
    if (cur_reg[id_left + i * height]) {
      shared_space[i] = i + 1; // store x cooridnates
    } else {
      shared_space[i] = 0; // if the pixel not on the region, set 0
    }
  }
  __syncthreads();
  // Continue after syncthreads
  // peform reduction to compute maximun x coordinates
  for (int s = (width + 1) / 2, e = width / 2; s > 0; s = (s + 1) / 2, e /= 2) {
    if (thread_id < e || thread_id ==0) {
      atomicMax(&shared_space[thread_id], shared_space[thread_id + s] );
    }
    __syncthreads();
    if (s == 1) break;
  }
  float max_center_x;
  if (thread_id == 0) {
    max_center_x = shared_space[0] / width_float;
  }
  //////////////////////////////////////////////////////////////////
  // reload shared_space to compute min_center_y
  for (int i = thread_id; i < height; i += blockDim.x) {
    if (cur_reg[id_top + i]) {
      shared_space[i] = i + 1; // store y cooridnates
    } else {
      shared_space[i] = height; // if the pixel not on the region, set maximun y
    }
  }
  __syncthreads();
  // Continue after syncthreads
  // peform reduction to compute minimun y coordinates
  for (int s = (height + 1) / 2, e = height / 2; s > 0; s = (s + 1) / 2, e /= 2) {
    if (thread_id < e || thread_id == 0) {
      atomicMin(&shared_space[thread_id], shared_space[thread_id + s] );
    }
       __syncthreads();
    if (s == 1) break;
  }
  float min_center_y;
  if (thread_id == 0) {
    min_center_y = shared_space[0] / height_float;
  }

  //////////////////////////////////////////////////////////////////
  // reload shared_space to compute max_center_y
  for (int i = thread_id; i < height; i += blockDim.x) {
    if (cur_reg[id_top + i]) {
      shared_space[i] = i + 1; // store y cooridnates
    } else {
      shared_space[i] = 0; // if the pixel not on the region, set to 0
    }
  }
  __syncthreads();
  // Continue after syncthreads
  // peform reduction to compute maximum y coordinates
  for (int s = (height + 1) / 2, e = height / 2; s > 0; s = (s + 1) / 2, e /= 2) {
    if (thread_id < e || thread_id == 0) {
      atomicMax(&shared_space[thread_id], shared_space[thread_id + s] );
    }
    __syncthreads();
    if (s == 1) break;
  }
  float max_center_y;
  if (thread_id == 0) {
    max_center_y = shared_space[0] / height_float;
    //}
    /////////////////////////////////////////////////////////////////////
    //if (thread_id == 0) {
  cur_geo_fea_float[0] = normalized_center_x; 
  cur_geo_fea_float[1] = normalized_center_y;
  cur_geo_fea_float[2] = normalized_max_x -normalized_min_x;
  cur_geo_fea_float[3] = normalized_max_y - normalized_min_y;
  cur_geo_fea_float[4] = (normalized_max_x - normalized_min_x) / (normalized_max_y - normalized_min_y);

  cur_geo_fea_float[5] = max_center_x - min_center_x;
  cur_geo_fea_float[6] = max_center_y - min_center_y;
  //cur_geo_fea_float[5] = min_center_y;
  //cur_geo_fea_float[6] = max_center_y; 
  float reg_area = cur_geo_fea_float[7];
  float normalized_reg_area = reg_area / (height_float * width_float);
  cur_geo_fea_float[7] = normalized_reg_area;
  cur_geo_fea_float[8] = normalized_reg_area / (normalized_max_x - normalized_min_x) / (normalized_max_y - normalized_min_y) ;
}
}
// central x, y, bounding box width, height, aspect ratio, major_length, minor_length, area, area/box, 


/**
 * Host routine
 */

//void compute_hist(const int *h_Qfea,const float *h_fea, const bool *h_lsm, const bool *h_Rlsm, const bool *h_reg, const int width, const int height, const int num_reg, const int num_bin, const int num_fea, const int num_ls_fea, unsigned int *h_hist, float *Lfea, float *LRfea ) {
void ComputeFeature(const int *host_qualified_fea, const float *host_fea, const bool *host_local_map, const bool *host_refined_local_map, const bool *host_reg, const int width, const int height, const int num_reg, const int num_bin, const int num_fea, const int num_local_fea, const float local_map_sum, const float refined_local_map_sum, float *host_hist, float *host_fea_mean, float *host_fea_var,  float* host_local_fea, float* host_refined_local_fea, float* host_geo_fea) {
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  size_t size_qualified_fea = width * height * num_fea * sizeof(int);
  size_t size_fea = width * height * COLOR_CHANNEL * num_fea * sizeof(float);
  size_t size_local_map = width * height * sizeof(bool);
  size_t size_refined_local_map = width * height * sizeof(bool);
  size_t size_reg = width * height * num_reg * sizeof(bool);
  size_t size_hist = num_bin * num_reg * num_fea * sizeof(float);
  size_t size_fea_mean = num_fea * COLOR_CHANNEL * num_reg * sizeof(float);
  size_t size_fea_var = num_fea * COLOR_CHANNEL * num_reg * sizeof(float);
  size_t size_local_fea = num_local_fea * num_reg * sizeof(float);
  size_t size_refined_local_fea = num_local_fea * num_reg * sizeof(float);
  size_t size_reg_area = num_reg * sizeof(float);
  //size_t size_local_fea_space = num_local_fea * num_reg * sizeof(float);
  size_t size_shared_hist = num_bin * sizeof(float);
  size_t size_shared_mean = num_fea * COLOR_CHANNEL * sizeof(float);
  size_t size_shared_var = num_fea * COLOR_CHANNEL * sizeof(float);
  size_t size_geo_fea = num_reg * NUM_GEO_FEA * sizeof(float);

  //  printf("cuda hist size: %d * %d * %d", num_bin, num_fea, num_reg);
  // Allocate the device_qualified_fea
  int *device_qualified_fea = NULL;
  AllocateDevice(qualified_fea);
  float *device_fea;
  AllocateDevice(fea);
  bool *device_local_map = NULL;
  AllocateDevice(local_map);
  bool *device_refined_local_map = NULL;
  AllocateDevice(refined_local_map);
  bool *device_reg = NULL;
  AllocateDevice(reg);
  float *device_hist = NULL;
  AllocateDevice(hist);
  float *device_fea_mean = NULL;
  AllocateDevice(fea_mean);
  float *device_fea_var = NULL;
  AllocateDevice(fea_var);
  float *device_local_fea = NULL;
  AllocateDevice(local_fea);
  float *device_refined_local_fea = NULL;
  AllocateDevice(refined_local_fea);
  float *device_reg_area = NULL;
  AllocateDevice(reg_area);
 // float *device_local_fea_space = NULL;
 // AllocateDevice(local_fea_space);
  float *device_geo_fea = NULL;
  AllocateDevice(geo_fea);

  CopyHosttoDevice(qualified_fea);
  CopyHosttoDevice(fea);
  CopyHosttoDevice(local_map);
  CopyHosttoDevice(refined_local_map);
  CopyHosttoDevice(reg);
  //CopyHosttoDevice(hist);
  //CopyHosttoDevice(fea_mean);
  //CopyHosttoDevice(fea_var);
  // CopyHosttoDevice(local_fea);
  //CopyHosttoDevice(refined_local_fea);

  SetDevice(hist, 0);
  SetDevice(fea_mean, 0);
  SetDevice(fea_var, 0);
  SetDevice(local_fea, 0);
  SetDevice(refined_local_fea, 0);
 // SetDevice(local_fea_space, 0);
  SetDevice(reg_area, 0);
  SetDevice(geo_fea, 0);


  // Launch the compute_local_saliency CUDA Kernel for local saliency map
  int threads_per_block = 256;
  int blocks_per_reg = (width * height + threads_per_block - 1) / threads_per_block;
  int blocks_per_grid = num_reg * blocks_per_reg;
  compute_local_saliency<<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>(device_local_map, device_reg, true, width, height, num_reg, num_local_fea, local_map_sum, blocks_per_reg, device_local_fea, device_reg_area);
  CUDA_CHECK(err);
  blocks_per_grid = (num_reg + threads_per_block - 1) / threads_per_block;
  compute_pr<<<blocks_per_grid, threads_per_block >>>(device_local_fea, device_reg_area, local_map_sum, num_reg, num_local_fea);
  CUDA_CHECK(err);
  // Launch the compute_local_saliency CUDA Kernel for refined local saliency map
  blocks_per_grid = num_reg * blocks_per_reg;
  compute_local_saliency<<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>(device_refined_local_map, device_reg, false, width, height, num_reg, num_local_fea, refined_local_map_sum, blocks_per_reg, device_refined_local_fea, device_reg_area);
  CUDA_CHECK(err);
  blocks_per_grid = (num_reg + threads_per_block - 1) / threads_per_block;
  compute_pr<<<blocks_per_grid, threads_per_block >>>(device_refined_local_fea, device_reg_area, refined_local_map_sum, num_reg, num_local_fea);
  CUDA_CHECK(err);
  // Launch the compute_color_hist CUDA Kernel
  blocks_per_grid = num_reg * blocks_per_reg;
  dim3 grid_dim;
  grid_dim.x = blocks_per_grid;
  grid_dim.y = num_fea; 
  compute_color_hist<<<grid_dim, threads_per_block, size_shared_hist>>>(device_qualified_fea, device_reg, device_reg_area,  width, height, num_reg, num_fea, num_bin, blocks_per_reg, device_hist);
  // Launch the compute_color_mean CUDA Kernel
  compute_color_mean<<<blocks_per_grid, threads_per_block, size_shared_mean>>>(device_fea, device_reg, device_reg_area, width, height, num_reg, num_fea, blocks_per_reg, device_fea_mean);
  // Launch the compute_var CUDA Kernel 
  compute_color_var<<<blocks_per_grid, threads_per_block, size_shared_var>>>(device_fea, device_fea_mean, device_reg, device_reg_area,  width, height, num_reg, num_fea, blocks_per_reg, device_fea_var);
  // Launch the compute_geo_fea CUDA Kernel
  compute_geo_fea<<<blocks_per_grid, threads_per_block>>>(device_reg, device_reg_area, width, height, num_reg, blocks_per_reg, device_geo_fea);
  threads_per_block = max(height, width);
  size_t size_shared_space = max(height, width) * sizeof(int);
  process_geo_fea<<<num_reg, threads_per_block, size_shared_space>>>(device_geo_fea, device_reg, num_reg, width, height);
  // central x, y, bounding box width, height, aspect ratio, major_length, minor_length, area, area/box, 
  CopyDevicetoHost(hist);
  CopyDevicetoHost(fea_mean);
  CopyDevicetoHost(fea_var);
  CopyDevicetoHost(local_fea);
  CopyDevicetoHost(refined_local_fea);
  CopyDevicetoHost(geo_fea);
  // Free device global memory
  FreeDevice(qualified_fea);
  FreeDevice(fea);
  FreeDevice(local_map);
  FreeDevice(refined_local_map);
  FreeDevice(reg);
  FreeDevice(hist);
  FreeDevice(fea_mean);
  FreeDevice(fea_var);
  FreeDevice(local_fea);
  FreeDevice(refined_local_fea);
  FreeDevice(reg_area);
 // FreeDevice(local_fea_space);
  FreeDevice(geo_fea);
  // Free host memory
  // free(h_img);
  // free(h_reg);
  // free(h_hist);

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits
  //  err = hipDeviceReset();
  //
  //  if (err != hipSuccess)
  //  {
  //    fprintf(stderr, "Failed to deinitialize the device! error=%s\num_reg", hipGetErrorString(err));
  //    exit(EXIT_FAILURE);
  //  }

  return;
}

